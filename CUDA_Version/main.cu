
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <cmath>
#include <string>
#include <cstdlib>

#define N 1000
#define STEPS 300
#define DT 0.01f
#define G 0.5
// 6.67430e-11f

struct Body {
    float x, y;
    float vx, vy;
    float mass;
};

float computeSpeed(const Body& b) {
    return std::sqrt(b.vx * b.vx + b.vy * b.vy);
}

void initializeBodies(std::vector<Body>& bodies) {
    bodies.resize(N);
    for (int i = 0; i < N; ++i) {
        bodies[i].x = static_cast<float>(rand()) / RAND_MAX * 100.0f;
        bodies[i].y = static_cast<float>(rand()) / RAND_MAX * 100.0f;
        bodies[i].vx = 0.0f;
        bodies[i].vy = 0.0f;
        bodies[i].mass = static_cast<float>(rand()) / RAND_MAX * 1e3f + 1.0f;
    }
}

void saveStep(const std::vector<Body>& bodies, const std::string& folder, int step) {
    std::ofstream fout(folder + "/positions_step_" + std::to_string(step) + ".txt");
    if (!fout) {
        std::cerr << "Error opening file for writing: " << folder << std::endl;
        return;
    }
    for (const auto& b : bodies) {
        fout << b.x << " " << b.y << " " << computeSpeed(b) << "\n";
    }
}

void simulateCPU(std::vector<Body>& bodies, const std::string& folder) {
    std::string cmd = "mkdir -p " + folder;
    system(cmd.c_str());

    for (int step = 0; step < STEPS; ++step) {
        std::vector<Body> next = bodies;
        for (int i = 0; i < N; ++i) {
            float fx = 0, fy = 0;
            for (int j = 0; j < N; ++j) {
                if (i == j) continue;
                float dx = bodies[j].x - bodies[i].x;
                float dy = bodies[j].y - bodies[i].y;
                float distSqr = dx * dx + dy * dy + 1e-4f;
                float invDist = 1.0f / std::sqrt(distSqr);
                float invDist3 = invDist * invDist * invDist;
                float f = G * bodies[i].mass * bodies[j].mass * invDist3;
                fx += f * dx;
                fy += f * dy;
            }
            next[i].vx += DT * fx / bodies[i].mass;
            next[i].vy += DT * fy / bodies[i].mass;
            next[i].x += DT * next[i].vx;
            next[i].y += DT * next[i].vy;
        }
        bodies = next;
        saveStep(bodies, folder, step);

        if(step % 50 == 0)
            std::cout << "[CPU] Step " << step << " done\n";
    }
}

__global__ void updateBodies(Body* bodies, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float fx = 0, fy = 0;
    for (int j = 0; j < n; ++j) {
        if (i == j) continue;
        float dx = bodies[j].x - bodies[i].x;
        float dy = bodies[j].y - bodies[i].y;
        float distSqr = dx * dx + dy * dy + 1e-4f;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;
        float f = G * bodies[i].mass * bodies[j].mass * invDist3;
        fx += f * dx;
        fy += f * dy;
    }

    bodies[i].vx += DT * fx / bodies[i].mass;
    bodies[i].vy += DT * fy / bodies[i].mass;
    bodies[i].x += DT * bodies[i].vx;
    bodies[i].y += DT * bodies[i].vy;
}

inline void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

void simulateGPU(std::vector<Body>& bodies, const std::string& folder) {
    std::string cmd = "mkdir -p " + folder;
    system(cmd.c_str());

    Body* d_bodies = nullptr;
    checkCuda(hipMalloc(&d_bodies, N * sizeof(Body)), "hipMalloc failed");
    checkCuda(hipMemcpy(d_bodies, bodies.data(), N * sizeof(Body), hipMemcpyHostToDevice), "hipMemcpy H2D failed");

    for (int step = 0; step < STEPS; ++step) {
        updateBodies<<<(N + 255) / 256, 256>>>(d_bodies, N);
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            std::cerr << "CUDA kernel error at step " << step << ": " << hipGetErrorString(err) << std::endl;
            break;
        }
        checkCuda(hipMemcpy(bodies.data(), d_bodies, N * sizeof(Body), hipMemcpyDeviceToHost), "hipMemcpy D2H failed");

        saveStep(bodies, folder, step);

        if(step % 50 == 0)
            std::cout << "[GPU] Step " << step << " done\n";
    }

    hipFree(d_bodies);
}

int main() {
    std::vector<Body> bodiesCPU, bodiesGPU;
    initializeBodies(bodiesCPU);
    bodiesGPU = bodiesCPU;

    std::cout << "Starting CPU simulation...\n";
    auto startCPU = std::chrono::high_resolution_clock::now();
    simulateCPU(bodiesCPU, "cpu_output");
    auto endCPU = std::chrono::high_resolution_clock::now();

    std::cout << "Starting GPU simulation...\n";
    auto startGPU = std::chrono::high_resolution_clock::now();
    simulateGPU(bodiesGPU, "gpu_output");
    auto endGPU = std::chrono::high_resolution_clock::now();

    auto cpuMs = std::chrono::duration_cast<std::chrono::milliseconds>(endCPU - startCPU).count();
    auto gpuMs = std::chrono::duration_cast<std::chrono::milliseconds>(endGPU - startGPU).count();

    std::cout << "CPU Time: " << cpuMs << " ms\n";
    std::cout << "GPU Time: " << gpuMs << " ms\n";

    return 0;
}
